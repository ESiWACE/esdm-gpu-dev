
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

hipfftHandle splan;
hipfftHandle dplan;

void sPlanCUFFT(int n, void *stream)
{
    hipfftPlan1d(&splan, n, HIPFFT_C2C, 1);
    hipfftSetStream(splan, (hipStream_t)stream);
}

void sExecCUFFT(float *sdata)
{
    hipfftExecC2C(splan, (hipfftComplex*)sdata, (hipfftComplex*)sdata, HIPFFT_FORWARD);
}

void sDestroyCUFFT()
{
    hipfftDestroy(splan);
}

void dPlanCUFFT(int n, void *stream)
{
    hipfftPlan1d(&dplan, n, HIPFFT_Z2Z, 1);
    hipfftSetStream(dplan, (hipStream_t)stream);
}

void dExecCUFFT(double *ddata)
{
    hipfftExecZ2Z(dplan, (hipfftDoubleComplex*)ddata, (hipfftDoubleComplex*)ddata, HIPFFT_FORWARD);
}

void dDestroyCUFFT()
{
    hipfftDestroy(dplan);
}

/*
void sLaunchCUFFT(float *sdata, int n, void *stream)
{
    cufftHandle plan;
    cufftPlan1d(&plan, n, CUFFT_C2C, 1);
    cufftSetStream(plan, (cudaStream_t)stream);
    cufftExecC2C(plan, (cufftComplex*)sdata, (cufftComplex*)sdata, CUFFT_FORWARD);
    cufftDestroy(plan);
}

void dLaunchCUFFT(double *ddata, int n, void *stream)
{
    cufftHandle plan;
    cufftPlan1d(&plan, n, CUFFT_Z2Z, 1);
    cufftSetStream(plan, (cudaStream_t)stream);
    cufftExecZ2Z(plan, (cufftDoubleComplex*)ddata, (cufftDoubleComplex*)ddata, CUFFT_FORWARD);
    cufftDestroy(plan);
}
*/
