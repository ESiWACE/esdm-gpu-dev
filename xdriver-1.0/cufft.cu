
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

hipfftHandle plan;
hipfftResult result;

// 1D FFT single precision ====================================================

void sPlan1dCUFFT(int n, void *stream) {
  result = hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftPlan1d failed: code = %i\n",result);
     return;
  }
  result = hipfftSetStream(plan, (hipStream_t)stream);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftSetStream failed: code = %i\n",result);
     return;
  }
}

// 2D FFT single precision ====================================================

void sPlan2dCUFFT(int nx, int ny, void *stream) {
  result = hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftPlan2d failed: code = %i\n",result);
     return;
  }
  result = hipfftSetStream(plan, (hipStream_t)stream);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftSetStream failed: code = %i\n",result);
     return;
  }
}

// single precision execute & destroy =========================================

void sExecCUFFT(float *sdata) {
  result = hipfftExecC2C(plan, (hipfftComplex*)sdata, (hipfftComplex*)sdata, HIPFFT_FORWARD);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftExecC2C failed: code = %i\n",result);
     return;
  }
}

void sDestroyCUFFT() {
  result = hipfftDestroy(plan);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftDestroy failed: code = %i\n",result);
     return;
  }
}

// 1D FFT double precision ====================================================

void dPlan1dCUFFT(int n, void *stream) {
  result = hipfftPlan1d(&plan, n, HIPFFT_Z2Z, 1);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftPlan1d failed: code = %i\n",result);
     return;
  }
  result = hipfftSetStream(plan, (hipStream_t)stream);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftSetStream failed: code = %i\n",result);
     return;
  }
}

// 2D FFT double precision ====================================================

void dPlan2dCUFFT(int nx, int ny, void *stream) {
  result = hipfftPlan2d(&plan, nx, ny, HIPFFT_Z2Z);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftPlan2d failed: code = %i\n",result);
     return;
  }
  result = hipfftSetStream(plan, (hipStream_t)stream);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftSetStream failed: code = %i\n",result);
     return;
  }
}

// double precision execute & destroy =========================================

void dExecCUFFT(double *ddata) {
  result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)ddata, (hipfftDoubleComplex*)ddata, HIPFFT_FORWARD);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftExecZ2Z failed: code = %i\n",result);
     return;
  }
}

void dDestroyCUFFT() {
  result = hipfftDestroy(plan);
  if (result!=HIPFFT_SUCCESS) { 
     printf ("Error: hipfftDestroy failed: code = %i\n",result);
     return;
  }
}

/*
// Original versions from the OLCF website
// https://www.olcf.ornl.gov/tutorials/mixing-openacc-with-gpu-libraries/
//
void sLaunchCUFFT(float *sdata, int n, void *stream) {
  cufftHandle plan;
  cufftPlan1d(&plan, n, CUFFT_C2C, 1);
  cufftSetStream(plan, (cudaStream_t)stream);
  cufftExecC2C(plan, (cufftComplex*)sdata, (cufftComplex*)sdata, CUFFT_FORWARD);
  cufftDestroy(plan);
}

void dLaunchCUFFT(double *ddata, int n, void *stream) {
  cufftHandle plan;
  cufftPlan1d(&plan, n, CUFFT_Z2Z, 1);
  cufftSetStream(plan, (cudaStream_t)stream);
  cufftExecZ2Z(plan, (cufftDoubleComplex*)ddata, (cufftDoubleComplex*)ddata, CUFFT_FORWARD);
  cufftDestroy(plan);
}
*/
